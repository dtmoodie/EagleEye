#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>
#include <opencv2/cudev.hpp>
#include <opencv2/core/matx.hpp>
#include <Aquila/utilities/thrust/thrust_interop.hpp>
#include <thrust/sequence.h>
#include <thrust/system/cuda/execution_policy.h>
namespace cv
{
namespace cuda
{
void histogram(const cv::cuda::GpuMat& in, cv::cuda::GpuMat& bins, cv::cuda::GpuMat& histogram,
               float min = 0, float max = 256,
               cv::cuda::Stream& stream = cv::cuda::Stream::Null());
}
}
template<typename T>
__host__ __device__ inline T* binary_search_approx(T *const begin, T * end, T value)
{
    T* q;
    if(begin >= end)
    {
        return end;
    }
    //q = (begin + end) / 2;
    q = begin + (end - begin) / 2;
    if(value == *q)
    {
        return q;
    }else if(value > *q)
    {
        return binary_search_approx(q + 1, end, value);
    }else if(value < *q)
    {
        return binary_search_approx(begin, q - 1, value);
    }
}


template<typename T, int N>
__global__ void histogram_kernel(const cv::cuda::PtrStepSz<cv::Vec<T, N>> input,
                                 const cv::cuda::PtrStepSz<float> bins,
                                 int* histogram, float min_value, float step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;
    int t = threadIdx.x + threadIdx.y * blockDim.x;

    int nt = blockDim.x * blockDim.y;

    const int num_bins = bins.cols;

    extern __shared__ int smem[];

    for (int i = t; i < N * num_bins + N; i += nt)
        smem[i] = 0;

    __syncthreads();

    for (int row = y; row < input.rows; row += ny)
    {
        for (int col = x; col < input.cols; col += nx)
        {
#pragma unroll
            for(int c = 0; c < N; ++c)
            {
                float val = input(row, col).val[c];
                // calculate bin based on min and step
                int idx = (val - min_value) / step;
                idx = max(0,min(bins.cols, idx));
                atomicAdd(&smem[idx * N + c], 1);
                //float* bin = binary_search_approx<float>(bins.data, bins.data + bins.cols, float(val));
                //int dist = bin - bins.data;
                //atomicAdd(&smem[dist * N + c], 1);
            }
        }
    }
      __syncthreads();

    for (int i = t; i < num_bins; i += nt) {
#pragma unroll
        for(int c = 0; c < N; ++c)
        {
            atomicAdd(histogram + i * N + c, smem[i * N + c]);
        }
    }
}

template<int N>
__global__ void histogram_kernel_uchar(const cv::cuda::PtrStepSz<cv::Vec<uchar, N>> input,
                                 int* histogram)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;
    int t = threadIdx.x + threadIdx.y * blockDim.x;

    int nt = blockDim.x * blockDim.y;

    const int num_bins = 256;

    //extern __shared__ int smem[N * num_bins + N];
    extern __shared__ int smem[];

    for (int i = t; i < N * num_bins + N; i += nt)
        smem[i] = 0;

    __syncthreads();

    for (int row = y; row < input.rows; row += ny)
    {
        for (int col = x; col < input.cols; col += nx)
        {
#pragma unroll
            for(int c = 0; c < N; ++c)
            {
                uchar val = input(row, col).val[c];
                atomicAdd(&smem[val* N + c], 1);
            }
        }
    }
      __syncthreads();


    for (int i = t; i < num_bins; i += nt) {
#pragma unroll
        for(int c = 0; c < N; ++c)
        {
            atomicAdd(histogram + i * N + c, smem[i * N + c]);
        }
    }
}

template<class T, int N>
void launch(const cv::cuda::GpuMat& in, cv::cuda::GpuMat& bins, cv::cuda::GpuMat& hist, cv::cuda::Stream& stream, float min, float step)
{
    dim3 block(16, 16);
    dim3 grid(cv::cudev::divUp(in.cols,16), cv::cudev::divUp(in.rows, 16));
    histogram_kernel<T,N><<<grid, block, bins.cols * N + N,
            cv::cuda::StreamAccessor::getStream(stream)>>>(
                in, bins, (int*)hist.data, min, step);
}
template<int N>
void launch_uchar(const cv::cuda::GpuMat& in, cv::cuda::GpuMat& bins, cv::cuda::GpuMat& hist, cv::cuda::Stream& stream, float min, float step)
{
    CV_Assert(in.depth() == CV_8U);
    CV_Assert(in.channels() == N);
    CV_Assert(hist.cols == 256 && hist.rows == 1 && hist.depth() == CV_32S && hist.channels() == N);
    dim3 block(16, 16);
    dim3 grid(cv::cudev::divUp(in.cols,16), cv::cudev::divUp(in.rows, 16));
    histogram_kernel_uchar<N><<<grid, block, (256 * N + N) * sizeof(int),
            cv::cuda::StreamAccessor::getStream(stream)>>>(
                in, (int*)hist.data);
}

void cv::cuda::histogram(const cv::cuda::GpuMat& in, cv::cuda::GpuMat& bins, cv::cuda::GpuMat& histogram,
                         float min, float max,
                         cv::cuda::Stream& stream)
{
    typedef void(*func_t)(const cv::cuda::GpuMat& in, cv::cuda::GpuMat& bins, cv::cuda::GpuMat& hist, cv::cuda::Stream& stream, float min, float step);
    int size = 1000;
    float step = 1;
    if(in.depth() == CV_8U)
    {
        size = 256;
        min = 0;
        max = 256;
    }
    if(bins.empty() && in.depth() != CV_8U)
    {
        bins.create(1, size, CV_32F);
        step = (max - min) / float(size);
        thrust::device_ptr<float> ptr = thrust::device_pointer_cast((float*)bins.data);
        thrust::sequence(thrust::system::cuda::par.on(cv::cuda::StreamAccessor::getStream(stream)),ptr, ptr + size, min, step);
    }
    histogram.create(1, size, CV_MAKE_TYPE(CV_32S, in.channels()));
    histogram.setTo(cv::Scalar::all(0), stream);
    func_t funcs[4][7] =
    {
        {launch_uchar<1>, 0, launch<ushort, 1>, 0, 0, 0, 0},
        {launch_uchar<2>, 0, launch<ushort, 2>, 0, 0, 0, 0},
        {launch_uchar<3>, 0, launch<ushort, 3>, 0, 0, 0, 0},
        {launch_uchar<4>, 0, launch<ushort, 4>, 0, 0, 0, 0}
    };
    CV_Assert(funcs[in.channels() - 1][in.depth()]);

    funcs[in.channels() - 1][in.depth()](in, bins, histogram, stream, min, step);
}
